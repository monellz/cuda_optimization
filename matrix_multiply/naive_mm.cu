#include "hip/hip_runtime.h"
#include <iostream>
#include "utils.h"
using namespace std;


#define N 32


__global__ void naive_mm(float* ma, float* mb, float* mc, int n) {
    for (int i = 0 + threadIdx.x; i < n; i += blockDim.x) {
        for (int j = 0 + threadIdx.y; j < n; j += threadIdx.y) {
            for (int k = 0; k < n; ++k) {
                mc[i * n + j] += ma[i * n + k] * mb[k * n + j];
            }
        }
    }
}


int main(int argc, char** argv) {
    float* ha = new float[N * N];
    float* hb = new float[N * N];
    float* hc = new float[N * N];

    float *da, *db, *dc;


    for (int i = 0; i < N * N; ++i) ha[i] = hb[i] = i;

    CUDA_CALL(hipMalloc(&da, sizeof(float) * N * N));
    CUDA_CALL(hipMalloc(&db, sizeof(float) * N * N));
    CUDA_CALL(hipMalloc(&dc, sizeof(float) * N * N));

    CUDA_CALL(hipMemcpy(da, ha, sizeof(float) * N * N, hipMemcpyHostToDevice));
    CUDA_CALL(hipMemcpy(db, hb, sizeof(float) * N * N, hipMemcpyHostToDevice));

    float time;
    hipEvent_t start, stop;
    CUDA_CALL(hipEventCreate(&start));
    CUDA_CALL(hipEventCreate(&stop));

    CUDA_CALL(hipEventRecord(start, 0));
    naive_mm<<<1, 1024>>>(da, db, dc, N);
    CUDA_CALL(hipEventRecord(stop, 0));
    CUDA_CALL(hipEventSynchronize(stop));
    CUDA_CALL(hipEventElapsedTime(&time, start, stop));
    printf("Time: %f ms\n", time);
    
    CUDA_CALL(hipMemcpy(hc, dc, sizeof(float) * N * N, hipMemcpyDeviceToHost));

    CUDA_CALL(hipFree(da));
    CUDA_CALL(hipFree(db));
    CUDA_CALL(hipFree(dc));

    delete[] ha;
    delete[] hb;
    delete[] hc;
}